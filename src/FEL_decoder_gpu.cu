#include "hip/hip_runtime.h"
#include "FEL_decoder_gpu.h"

__device__
void flexibleFractalDecoding( float* original, float* decoded, int nBin, int flip, int shift)
{
	int m = 0;
	if (flip == 0)
	{
		for (int i = 0; i < nBin; i++)
		{
			m = i + shift;
			if (m >= nBin) 
				m = m - nBin; 
		
			decoded[m] = original[i];
		}// end for
	}
	else 
	{
		float temp[64];
		for (int i = 0; i < nBin; i++) 
		{
			temp[i] = original[nBin - 1 - i];
		}
		
		for (int i = 0; i < nBin; i++) 
		{
			m = i + shift;
			if (m >= nBin)
				m = m - nBin;
			decoded[m] = temp[i];
		}		
	}
}

__global__ void
d_divideBlock( int divPar)
{
	// TODO: WARNING! here we could not use x in this function, otherwise cuda kernel will have error, the problem occurs when we use span[i].low.y
	//int divPar = 30;
	// NOTE: here we assume the input volume data is nDimension*nDimension*nDimension, which means, we didin't consider dataset such as isabel 500x500x100
	// divPar is the size of each block specified by user
	//printf("start d_divideBlock!\n");
	if (divPar > nDimension) {
		printf("wrong divPar! %d\n", divPar);
	}

	int n = 0; // number of blocks divided in each dimension, so that n*n*n is the total number of blocks	
	int2 spanX[flexNBin];	// for each dimension, record the spanLow and spanHigh, x is spanLow, y is spanHigh
	int2 spanY[flexNBin];
	int2 spanZ[flexNBin];
	if (nDimension % divPar != 0) {
		n = nDimension / divPar + 1;
		for (int i = 0; i < n; i++) {
			spanX[i].x = 1 + i * divPar;
			spanY[i].x = 1 + i * divPar;
			spanZ[i].x = 1 + i * divPar;
			if (i != n - 1) {
				spanX[i].y = (i + 1) * divPar;
				spanY[i].y = (i + 1) * divPar;
				spanZ[i].y = (i + 1) * divPar;
			}
			else {
				spanX[i].y = flexNBin;
				spanY[i].y = flexNBin;
				spanZ[i].y = flexNBin;
			}
		}
	}
	else {
		n = nDimension / divPar;
		for (int i = 0; i < n; i++) {
			spanX[i].x = 1 + i * divPar;
			spanY[i].x = 1 + i * divPar;
			spanZ[i].x = 1 + i * divPar;
			spanX[i].y = (i + 1) * divPar;
			spanY[i].y = (i + 1) * divPar;
			spanZ[i].y = (i + 1) * divPar;
		}
	}
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			for (int k = 0; k < n; k++)
			{
				flexBlock[i*n*n + j*n + k].low.x = spanX[i].x;
				flexBlock[i*n*n + j*n + k].low.y = spanY[j].x;
				flexBlock[i*n*n + j*n + k].low.z = spanZ[k].x;
				flexBlock[i*n*n + j*n + k].high.x = spanX[i].y;
				flexBlock[i*n*n + j*n + k].high.y = spanY[j].y;
				flexBlock[i*n*n + j*n + k].high.z = spanZ[k].y;
				//printf("blockLow(%2d, %2d, %2d), blockHigh(%2d, %2d, %2d)\n", flexBlock[i*n*n + j*n + k].low.x, flexBlock[i*n*n + j*n + k].low.y, flexBlock[i*n*n + j*n + k].low.z, flexBlock[i*n*n + j*n + k].high.x, flexBlock[i*n*n + j*n + k].high.y, flexBlock[i*n*n + j*n + k].high.z);
			}
		}
	}
	nFlexBlock = n * n * n;
	nFlexBlockX = n;
	nFlexBlockY = n;
	nFlexBlockZ = n;
	printf("nFlexBlock: %d\n", nFlexBlock);
	// TODO: call d_decompose to decompose each span and query the histogram
	//printf("finsh d_divideBlock!\n");
}

__global__ void
d_sumSanHistogram(int n) {
	int cId = blockIdx.x;

	// initialize cornerSumHistogram
	for (int i = 0; i < flexNBin; i++) {
		cornerSumHistogram[cId][i] = 0;
	}

	int totalWeight = 0;
	for (int i = 0; i < nSubSpan[cId]; i++) {
		Span currentSpan = subSpan[cId][i];
		int weight = (currentSpan.high.x - currentSpan.low.x + 1) * (currentSpan.high.y - currentSpan.low.y + 1) * (currentSpan.high.z - currentSpan.low.z + 1);
		for (int j = 0; j < flexNBin; j++) {
			cornerSumHistogram[cId][j] += cornerHistogram[cId][i][j] * weight;
			//if (n == 5 && cornerHistogram[cId][i][j] > 0.000001) {
			//	printf("nSubSpan[%2d], binId[%2d], freq: %f\n", i, j, cornerHistogram[cId][i][j]);
			//}
			//if (n == 5) {
			//	printf("cornerSumHistogram[%2d][%2d]: %f\n", cId, j, cornerSumHistogram[cId][j]);
			//}
		}
		totalWeight += weight;
	}

	/* old normalize method
	// normalize cornerSumHistogram
	float total = 0;
	for (int i = 0; i < flexNBin; i++) {
		total += cornerSumHistogram[cId][i];
	}
	printf("total[%2d]: %f\n", cId, total);
	for (int i = 0; i < flexNBin; i++) {
		cornerSumHistogram[cId][i] = cornerSumHistogram[cId][i] / total;
		if (cornerSumHistogram[cId][i] < 0 || cornerSumHistogram[cId][i] > 1) {
			printf("wrong cornerSumHistogram[%d][%d]: %f\n", cId, i, cornerSumHistogram[cId][i]);
		}
	}*/

	//if (n == 13) {
	//	printf("\nhistogram of block 13:\n");
	//}
	/*
	float total = 0;
	for (int i = 0; i < flexNBin; i++) {
		cornerSumHistogram[cId][i] = cornerSumHistogram[cId][i] / totalWeight;
		//if (n == 13) {
		//	printf("%1.6f\t", cornerSumHistogram[cId][i]);
		//}
		total += cornerSumHistogram[cId][i];
	}
	if (total < 0.999999 || total > 1.000001) {
		printf("n = %2d, corner = %d, total = %f\n", n, cId, total);
	}*/
}

__global__ void
d_computeBlock(int n) {
	// n is the index of current block
	float blockHistogram[flexNBin];
	
	//if (n == 13) {
	//	Span cs = flexBlock[n];
	//	printf("histogram of block 13: spanLow(%d, %d, %d), spanHigh(%d, %d, %d)\n", cs.low.x, cs.low.y, cs.low.z, cs.high.x, cs.high.y, cs.high.z);
	//}
	for (int s = 0; s < flexNBin; s++) {
		
		/*
		// test if the bin values are 0
		if (cornerSumHistogram[0][s] > 100000) { printf("cornerSumHistogram[0][%2d] = %f\n", s, cornerSumHistogram[0][s]); }
		if (cornerSumHistogram[1][s] > 100000) { printf("cornerSumHistogram[1][%2d] = %f\n", s, cornerSumHistogram[1][s]); }
		if (cornerSumHistogram[2][s] > 100000) { printf("cornerSumHistogram[2][%2d] = %f\n", s, cornerSumHistogram[2][s]); }
		if (cornerSumHistogram[3][s] > 100000) { printf("cornerSumHistogram[3][%2d] = %f\n", s, cornerSumHistogram[3][s]); }
		if (cornerSumHistogram[4][s] > 100000) { printf("cornerSumHistogram[4][%2d] = %f\n", s, cornerSumHistogram[4][s]); }
		if (cornerSumHistogram[5][s] > 100000) { printf("cornerSumHistogram[5][%2d] = %f\n", s, cornerSumHistogram[5][s]); }
		if (cornerSumHistogram[6][s] > 100000) { printf("cornerSumHistogram[6][%2d] = %f\n", s, cornerSumHistogram[6][s]); }
		if (cornerSumHistogram[7][s] > 100000) { printf("cornerSumHistogram[7][%2d] = %f\n", s, cornerSumHistogram[7][s]); }
		*/

		blockHistogram[s] = cornerSumHistogram[0][s] + cornerSumHistogram[5][s] + cornerSumHistogram[2][s] + cornerSumHistogram[7][s] - cornerSumHistogram[1][s] - cornerSumHistogram[4][s] - cornerSumHistogram[3][s] - cornerSumHistogram[6][s]; 
		if (blockHistogram[s] < 0) { blockHistogram[s] = 0; }	// NOTE: here we just make sure it has no negative value
		if (n == 5 && blockHistogram[s] > 0.000001) {
			printf("i = %2d, freq = %1.6f\n", s, blockHistogram[s]);
		}
	}

	// normalize blockHistogram
	Span currentSpan = flexBlock[n];
	//int weight = (currentSpan.high.x - currentSpan.low.x + 1) * (currentSpan.high.y - currentSpan.low.y + 1) * (currentSpan.high.z - currentSpan.low.z + 1);
	//if (n == 5) {
	//	printf("n = %2d, spanLow(%2d, %2d, %2d), spanHigh(%2d, %2d, %2d), weight = %d\n", n, currentSpan.low.x, currentSpan.low.y, currentSpan.low.z, currentSpan.high.x, currentSpan.high.y, currentSpan.high.z, weight);
	//}
	float totalBlockHistogram = 0;
	for (int s = 0; s < flexNBin; s++) {
		totalBlockHistogram += blockHistogram[s];
	}
	if (totalBlockHistogram <= 0) {
		printf("block %d: totalBlockHistogram = %f\n", n, totalBlockHistogram);
	}
	else {
		for (int s = 0; s < flexNBin; s++) {
			blockHistogram[s] = blockHistogram[s] / totalBlockHistogram;
			if (blockHistogram[s] < 0) { blockHistogram[s] = 0; }
			if (blockHistogram[s] > 1) { blockHistogram[s] = 1; }
			//if (n >= 24 && n < 27) {
			//	printf("%1.6f\n", blockHistogram[s]);
			//}
		}
	}


	//compute entropy
	float entropy = 0;
	for( int i = 0; i < flexNBin; i++ )
	{
		float probability = blockHistogram[i];
		//printf("%2d %d %f\t", n, i, probability);
		entropy += ( probability * ( probability <= 0 ? 0 : ( log( probability ) / log(2.0) ) ) );
	}

	// Change sign
	entropy = -entropy;

	// Normalize, if required
	entropy /= ( log( (float)flexNBin ) / log( 2.0f ) );

	flexBlockData[n].z = entropy;
	//printf("blockId = %2d, entropy = %f\n", n, entropy);
}

// d_queryBlockNew is to get subSpan of each corner
__global__ void
d_queryBlockNew(int blockNumber) {
	//printf("blockIdx: %d, threadIdx: %d\n", blockIdx.x, threadIdx.x);
	//int4 SpanLow = make_int4(15, 20, 25, 0); 
	//int4 SpanHigh = make_int4(21, 28, 33, 0);
	int4 SpanLow = flexBlock[blockNumber].low;
	int4 SpanHigh = flexBlock[blockNumber].high;
	int cornerId = blockIdx.x;
	//int spanId = threadIdx.x;
	switch (cornerId) {
	// 2: (0, 1, 0)		3: (1, 1, 0)
	// 0: (0, 0, 0)		1: (1, 0, 0)
	// 6: (0, 1, 1)		7: (1, 1, 1)
	// 4: (0, 0, 1)		5: (1, 0, 1)
	case 0:
		corner[0].x = (SpanLow.x < SpanHigh.x) ? SpanLow.x : SpanHigh.x;
		corner[0].y = (SpanLow.y < SpanHigh.y) ? SpanLow.y : SpanHigh.y;
		corner[0].z = (SpanLow.z < SpanHigh.z) ? SpanLow.z : SpanHigh.z;
		corner[0].w = 0;
		break;
	case 1:
		corner[1].x = (SpanLow.x > SpanHigh.x) ? SpanLow.x : SpanHigh.x;
		corner[1].y = (SpanLow.y < SpanHigh.y) ? SpanLow.y : SpanHigh.y;
		corner[1].z = (SpanLow.z < SpanHigh.z) ? SpanLow.z : SpanHigh.z;
		corner[1].w = 0;
		break;
	case 2:
		corner[2].x = (SpanLow.x < SpanHigh.x) ? SpanLow.x : SpanHigh.x;
		corner[2].y = (SpanLow.y > SpanHigh.y) ? SpanLow.y : SpanHigh.y;
		corner[2].z = (SpanLow.z < SpanHigh.z) ? SpanLow.z : SpanHigh.z;
		corner[2].w = 0;
		break;
	case 3:
		corner[3].x = (SpanLow.x > SpanHigh.x) ? SpanLow.x : SpanHigh.x;
		corner[3].y = (SpanLow.y > SpanHigh.y) ? SpanLow.y : SpanHigh.y;
		corner[3].z = (SpanLow.z < SpanHigh.z) ? SpanLow.z : SpanHigh.z;
		corner[3].w = 0;
		break;
	case 4:
		corner[4].x = (SpanLow.x < SpanHigh.x) ? SpanLow.x : SpanHigh.x;
		corner[4].y = (SpanLow.y < SpanHigh.y) ? SpanLow.y : SpanHigh.y;
		corner[4].z = (SpanLow.z > SpanHigh.z) ? SpanLow.z : SpanHigh.z;
		corner[4].w = 0;
		break;
	case 5:
		corner[5].x = (SpanLow.x > SpanHigh.x) ? SpanLow.x : SpanHigh.x;
		corner[5].y = (SpanLow.y < SpanHigh.y) ? SpanLow.y : SpanHigh.y;
		corner[5].z = (SpanLow.z > SpanHigh.z) ? SpanLow.z : SpanHigh.z;
		corner[5].w = 0;
		break;
	case 6:
		corner[6].x = (SpanLow.x < SpanHigh.x) ? SpanLow.x : SpanHigh.x;
		corner[6].y = (SpanLow.y > SpanHigh.y) ? SpanLow.y : SpanHigh.y;
		corner[6].z = (SpanLow.z > SpanHigh.z) ? SpanLow.z : SpanHigh.z;
		corner[6].w = 0;
		break;
	case 7:
		corner[7].x = (SpanLow.x > SpanHigh.x) ? SpanLow.x : SpanHigh.x;
		corner[7].y = (SpanLow.y > SpanHigh.y) ? SpanLow.y : SpanHigh.y;
		corner[7].z = (SpanLow.z > SpanHigh.z) ? SpanLow.z : SpanHigh.z;
		corner[7].w = 0;
		break;
	}
	//if (threadIdx.x == 0) {
	//	printf("corner[%d] %d %d %d %d\n", cornerId, corner[cornerId].x, corner[cornerId].y, corner[cornerId].z, corner[cornerId].w);
	//}
	int2 subSpanX[6];	// NOTE: here we hard code array size 6, because bin number is 64 = 2^6,
					// there are at most 6 sub spans in each dimension
					// subSpanX[i].x is the lower span, subSpanX[i].y is the upper span
	int2 subSpanY[6];
	int2 subSpanZ[6];
	//Span subSpan[6*6*6]; // at most 6*6*6 sub spans
	int nx = 0;			// count how many spans generated in each dimension
	int ny = 0;
	int nz = 0;
	int n = 0;	// total number of subSpans
	int x = corner[cornerId].x;
	int y = corner[cornerId].y;
	int z = corner[cornerId].z;
	//printf("corner[%d] %d %d %d\n", t, corners[t].x, corners[t].y, corners[t].z);
		
	for (int i = 0; i < 6; i++) {
		if ((x & (~(1 << i))) != x) {
			subSpanX[nx].y = x;
			x &= ~(1 << i);
			subSpanX[nx].x = x + 1;
			//printf("subSpanX[%d]: low %d, high %d\n", nx, subSpanX[nx].x, subSpanX[nx].y);
			nx++;
		}
		if (x == 0) { break; }
	}
	for (int i = 0; i < 6; i++) {
		if ((y & (~(1 << i))) != y) {
			subSpanY[ny].y = y;
			y &= ~(1 << i);
			subSpanY[ny].x = y + 1;
			//printf("subSpanY[%d]: low %d, high %d\n", ny, subSpanY[ny].x, subSpanY[ny].y);
			ny++;
		}
		if (y == 0) { break; }
	}
	for (int i = 0; i < 6; i++) {
		if ((z & (~(1 << i))) != z) {
			subSpanZ[nz].y = z;
			z &= ~(1 << i);
			subSpanZ[nz].x = z + 1;
			//printf("subSpanZ[%d]: low %d, high %d\n", nz, subSpanZ[nz].x, subSpanZ[nz].y);
			nz++;
		}
		if (z == 0) { break; }
	}
	n = nx * ny * nz;
	nSubSpan[cornerId] = n;
	for (int i = 0; i < nx; i++) {
		for (int j = 0; j < ny; j++) {
			for (int k = 0; k < nz; k++) {
				subSpan[cornerId][i*ny*nz + j*nz + k].low.x = subSpanX[i].x;	
				subSpan[cornerId][i*ny*nz + j*nz + k].low.y = subSpanY[j].x;
				subSpan[cornerId][i*ny*nz + j*nz + k].low.z = subSpanZ[k].x;
				subSpan[cornerId][i*ny*nz + j*nz + k].high.x = subSpanX[i].y;
				subSpan[cornerId][i*ny*nz + j*nz + k].high.y = subSpanY[j].y;
				subSpan[cornerId][i*ny*nz + j*nz + k].high.z = subSpanZ[k].y;
			}
		}
	}
	
	if (threadIdx.x == 0) {
		/*
		for (int i = 0; i < n; i++) {
			printf("subSpan[%2d]: low(%2d, %2d, %2d)  high(%2d, %2d, %2d)\n", i,
					subSpan[cornerId][i].low.x, subSpan[cornerId][i].low.y, subSpan[cornerId][i].low.z,
					subSpan[cornerId][i].high.x, subSpan[cornerId][i].high.y, subSpan[cornerId][i].high.z);
		}
		*/
	//	printf("nSubSpan[%d] (%2d, %2d, %2d) n = %d\n", cornerId, corner[cornerId].x, corner[cornerId].y, corner[cornerId].z, nSubSpan[cornerId]);
	}


	//float subSpanHistogram[flexNBin];	// this array stores the sum of all subSpan histograms of each corner
	//for (int countSSH = 0; countSSH < flexNBin; countSSH++) {
	//	subSpanHistogram[countSSH] = 0;
	//}
	// look up histogram of n subSpans
	//printf("spanLow %d %d %d, spanHigh %d %d %d, ", SpanLow.x, SpanLow.y, SpanLow.z, SpanHigh.x, SpanHigh.y, SpanHigh.z);
	//printf("current t %d, n %d, nx %d, ny %d, nz %d\n", t, n, nx, ny, nz);
}

// d_querySpanNew is to get histogram of each span
__global__ void
d_querySpanNew() {

	int cId = blockIdx.x;	// corner Id
	int sId = threadIdx.x;	// span Id
	if (sId >= nSubSpan[cId])
		return;

	Span currentSpan = subSpan[cId][sId];
	int4 currentFractal;	// current fractal encoded histogram
	int indexError = 0;		// index of NE in errorTexture
			
	// NOTE: Here we need to use a threshold to distinguish whether this span is fractal encoded or simple histogram
	//if (d_spanSize(currentSpan) >= 8) {
	if (1) 
	{	
		// look up fractal codebook
		int found = 0;
		for (int iz = 0; iz < nDimension/2; iz++) {
			for (int iy = 0; iy < nDimension; iy++) {
				for (int ix = 0; ix < nDimension; ix++) {	// NOTE: the fuel dataset size is 64x64x64, but half of them are encoded by fracal and half by simple histogram, so that each texture has 64x64x32 size
					int4 spanLow = tex3D(codebookSpanLowTex, ix, iy, iz);
					int4 spanHigh = tex3D(codebookSpanHighTex, ix, iy, iz);		
					if ((spanLow.x == currentSpan.low.x)
							&& (spanLow.y == currentSpan.low.y)
							&& (spanLow.z == currentSpan.low.z)
							&& (spanHigh.x == currentSpan.high.x)
							&& (spanHigh.y == currentSpan.high.y)
							&& (spanHigh.z == currentSpan.high.z)) {
						currentFractal = tex3D(flexibleCodebookTex, ix, iy, iz);
						// TODO: add NE
						//indexError = ix * iy * iz;
						// NOTE: how to index error list?
						indexError = ix + iy * nDimension + iz * nDimension * nDimension;
						//printf("cId %d, sId %2d, currentFractal: ID %3d, shift %2d, flip %d, NE %d\n", cId, sId, currentFractal.x, currentFractal.y, currentFractal.z, currentFractal.w);
						found = 1;
						break;
					}
				}
			}
		}// end look up fractal codebook
		if (found == 0) {
			printf("didn't find fractal: cId %d, sId %2d, spanLow(%2d, %2d, %2d), spanHigh(%2d, %2d, %2d)\n", cId, sId, currentSpan.low.x, currentSpan.low.y, currentSpan.low.z, currentSpan.high.x, currentSpan.high.y, currentSpan.high.z);
		}
		
		// fractal decoding
		// Flip (if needed)->Shift->merge errors->normalize the frequencies
		float originalTemplate[flexNBin];
		float currentTemplate[flexNBin];
		int templateId = currentFractal.x;
		// TODO: find out why next line makes program crush
		if (templateId < 0 || templateId > nTemplate) { printf("Error! templateID: %d\n", templateId); }
		int shift = currentFractal.y;
		int flipFlag = currentFractal.z;
		int NE = currentFractal.w;
		// TODO: find out why we cannot print NE
		if (NE < 0 || NE > flexNBin) {
			printf("Error NE!: %d\n", NE);
		}
		//float sumOT = 0;
		for (int i = 0; i < flexNBin; i++) {
		// TODO: find out why next line makes program crush
			originalTemplate[i] = tex2DLayered(flexibleTemplatesTex, (float) i, (float) templateId, 1);
			if (originalTemplate[i] < 0 || originalTemplate[i] > 1) {
				printf("Error! originalTemplate[%d]: %f\n", i, originalTemplate[i]);
			}
			//sumOT += originalTemplate[i];
		}
		//printf("cId %d, sId %2d, sumOT %f\n", cId, sId, sumOT);
		flexibleFractalDecoding( originalTemplate, currentTemplate, flexNBin, flipFlag, shift);
		for (int i = 0; i < flexNBin; i++) {
			if (currentTemplate[i] < 0 || currentTemplate[i] > 1) {
				printf("Error! currentTemplate[%d], %f\n", i, currentTemplate[i]);
			}
		}
		// add NE
		for (int i = 0; i < NE; i++) {
			// TODO: find out why we could not look up texture and use if printf command
			float2 error = tex2DLayered(flexibleErrorsbookTex, (float) i, (float)(indexError % 2048), (int) (indexError / 2048) );
			int errorIndex = (int)error.x;
			if (errorIndex < 0 || errorIndex > flexNBin) {
				printf("Error Index! %d\n", errorIndex);
			}
			float errorValue = error.y;
			if (errorValue < -1 || errorValue > 1) {
				printf("Error Value! %f\n", errorValue);
			}
			// TODO: find out why next line makes program crush
			currentTemplate[errorIndex] += errorValue;	// TODO: in this step, we get wrong value
			if (currentTemplate[errorIndex] < 0 ) { currentTemplate[errorIndex] = 0; }
		}
		// normalize currenTemplate
		float tempTotal = 0;
		for (int i = 0; i < flexNBin; i++) {
			tempTotal += currentTemplate[i];
		}
		for (int i = 0; i < flexNBin; i++) {
			currentTemplate[i] = currentTemplate[i] / tempTotal;
			if (currentTemplate[i] < 0 || currentTemplate[i] > 1) { 
				printf("Error! currentTemplate[%d]: %f\n", i, currentTemplate[i]); 
			}
		}

		/*
		// test currentTemplate
		float sumCT = 0;
		for (int i = 0; i < flexNBin; i++) {
			sumCT += currentTemplate[i];
		}
		if (sumCT > 1.000001 || sumCT < 0.999999) {
			printf("cId %d, sId %2d, sumCT: %f\n", cId, sId, sumCT);
		}*/
		

		// copy currentTemplate into cornerHistogram
		for (int countCopy = 0; countCopy < flexNBin; countCopy++) {
			cornerHistogram[cId][sId][countCopy] = currentTemplate[countCopy];
		}

		
		// test currentTemplate
		float sumCT = 0;
		for (int i = 0; i < flexNBin; i++) {
			sumCT += cornerHistogram[cId][sId][i];
		}
		if (sumCT > 1.000001 || sumCT < 0.999999) {
			printf("cId %d, sId %2d, sumCT: %f\n", cId, sId, sumCT);
		}

		// NOTE: we could not directly copy currentTemplate to cornerHistogram, beacuse they need synchronize
		// TODO: use shared memory
	}
	else {
		//look up simple codebook
		// look up simple histogram
		//printf("simple! cId %d, sId %2d, low(%2d, %2d, %2d), high(%2d, %2d, %2d)\n", cId, sId, currentSpan.low.x, currentSpan.low.y, currentSpan.low.z, currentSpan.high.x, currentSpan.high.y, currentSpan.high.z);
		
		// NOTE: when we are doing bitwise operation, we assume range (1 ~ 64), but the simple span data range is (0 ~ 63)
		// NOTE: but the fractal span range is still (1 ~ 64)
		currentSpan.low.x -= 1;
		currentSpan.low.y -= 1;
		currentSpan.low.z -= 1;
		currentSpan.high.x -= 1;
		currentSpan.high.y -= 1;
		currentSpan.high.z -= 1;
		
		int currentSimpleCount = 0;
		int indexSimpleHistogram = 0;

		float currentSimpleHistogram[flexNBin];
		// initialize
		for (int i = 0; i < flexNBin; i++) {
			currentSimpleHistogram[i] = 0;
		}
		
		int found = 0;
		
		for (int iz = 0; iz < nDimension/2; iz++) {
			for (int iy = 0; iy < nDimension; iy++) {
				for (int ix = 0; ix < nDimension; ix++) {
					int4 spanLow = tex3D(simpleSpanLowTex, ix, iy, iz);
					int4 spanHigh = tex3D(simpleSpanHighTex, ix, iy, iz);
					//if (ix < 5 && iy < 5 && iz < 5) {
					//	printf("%d %d %d, spanLow(%d, %d, %d) spanHigh(%d, %d, %d)\n", ix, iy, iz, spanLow.x, spanLow.y, spanLow.z, spanHigh.x, spanHigh.y, spanHigh.z);
					//}
					if (spanLow.x == currentSpan.low.x
							&& spanLow.y == currentSpan.low.y
							&& spanLow.z == currentSpan.low.z
							&& spanHigh.x == currentSpan.high.x
							&& spanHigh.y == currentSpan.high.y
							&& spanHigh.z == currentSpan.high.z) {
						currentSimpleCount = tex3D(simpleCountTex, ix, iy, iz);
						//indexSimpleHistogram = ix * iy * iz;
						// NOTE: how to index simple histogram?
						indexSimpleHistogram = ix + iy * nDimension + iz * nDimension * nDimension;
						//printf("cId %d, sId %2d, Count %2d, index %d\n", cId, sId, currentSimpleCount, indexSimpleHistogram);
						found = 1;
						break;
					}
				}
			}
		}// end looking up simpleHistogram
		if (found == 0) {
			printf("didn't find simple! cId %d, sId %2d, simpleLow(%2d, %2d, %2d), simpleHigh(%2d, %2d, %2d)\n", cId, sId, currentSpan.low.x, currentSpan.low.y, currentSpan.low.z, currentSpan.high.x, currentSpan.high.y, currentSpan.high.z);
		}
		for (int ih = 0; ih < currentSimpleCount; ih++) {
			float2 current = tex2DLayered(simpleHistogramTex, ih, indexSimpleHistogram % 2048, indexSimpleHistogram / 2048);
			currentSimpleHistogram[(int)current.x] = current.y;
			//printf("cId %d, sId %d, bin %d, freq %f\n", cId, sId, current.x, current.y);
		}
		for (int countCopy = 0; countCopy < flexNBin; countCopy++) {
			cornerHistogram[cId][sId][countCopy] = currentSimpleHistogram[countCopy];
		}
		
		
		// test currentTemplate
		float sumCT = 0;
		for (int i = 0; i < flexNBin; i++) {
			sumCT += cornerHistogram[cId][sId][i];
		}
		if (sumCT > 1.000001 || sumCT < 0.999999) {
			printf("cId %d, sId %2d, sumCT: %f\n", cId, sId, sumCT);
		}
		
	}// end look up simple codebook
}

__global__ void
d_clearCornerHistogram() {
	int cId = blockIdx.x;
	int tId = threadIdx.x;
//	cornerHistogram[cId][tId] = 0;
}

void dataProcessing() {
	//d_queryBlock<<<1, 1>>>(make_int4(15, 20, 25, 0), make_int4(15, 20, 25, 0));
	//d_queryBlockNew<<<8, 1>>>(make_int4(15, 20, 25, 0), make_int4(21, 28, 33, 0));
	//d_queryBlockNew<<<8, 1>>>(3);
	//d_querySpanNew<<<8, 216>>>();
	d_divideBlock<<<1, 1>>>(30);
	int h_nFlexBlock;	// how many blocks in this volume
	hipMemcpyFromSymbol(&h_nFlexBlock, HIP_SYMBOL(nFlexBlock), sizeof(int));
	printf("h_nFlexBlock: %d\n", h_nFlexBlock);
	
	for (int i = 0; i < h_nFlexBlock; i++) {
		//d_clearCornerHistogram<<<8, flexNBin>>>();
		d_queryBlockNew<<<8, 1>>>(i);
		d_querySpanNew<<<8, 216>>>();
		d_sumSanHistogram<<<8, 1>>>(i);
		//d_normalizeCorner<<<8, 216>>>();
		d_computeBlock<<<1, 1>>>(i);
		//TODO: store the result into an array, and pass that array into a texture
	}
	//d_testSimpleSum<<<1, 1>>>();
	hipDeviceSynchronize();
	//d_testEntropy<<<1, 1>>>();
	//bindToTex();
	//d_testTexRef<<<1, 1>>>();
	//bindToTexObj();
	//d_testTexObj<<<1, 1>>>(flexBlockTexObj);

	//bindToSurfObj();
	//d_testSurfObj<<<1, 1>>>(flexBlockSurfObj);

	//d_testMallocMem<<<1, 1>>>();
	//d_testTd<<<3, 3>>>();
	//d_testSharedMemory<<<1, 4>>>();
	//d_testPrintShared<<<1, 1>>>();
	//d_showEntropy<<<1, h_nFlexBlock>>>();
	//d_testSimpleSpan<<<64, 64>>>();
}


void 
initCuda( int4 *h_codebookSpanLow, int4 *h_codebookSpanHigh,
		  int4 *h_flexibleCodebook, 
		  float2 *h_flexibleErrorsbook, 
		  int4 *h_simpleLow, int4 *h_simpleHigh, int *h_simpleCount,
		  float2 *h_simpleHistogram, float *h_flexibleTemplates )
{
	// create a volume data in which each voxel has a index number
	// which can be used to lookup the histogram location
	//int volume[nBlocks];
	//for (int i = 0; i < nBlocks; i++)
	//	volume[i] = i;
	
    ////////////////////////////////////////////
    // flexible block size
    ////////////////////////////////////////////

    // create 3D array for codebookSpanLow
	hipChannelFormatDesc channelDescCodebookSpanLow = hipCreateChannelDesc<SpanType>();
    checkCudaErrors(hipMalloc3DArray(&d_codebookSpanLowArray, &channelDescCodebookSpanLow, flexibleVolumeSize));

    // create 3D array for codebookSpanHigh
    hipChannelFormatDesc channelDescCodebookSpanHigh = hipCreateChannelDesc<SpanType>();
    checkCudaErrors(hipMalloc3DArray(&d_codebookSpanHighArray, &channelDescCodebookSpanHigh, flexibleVolumeSize));

    // create 3D array for flexibleCodebook
    hipChannelFormatDesc channelDescFlexibleCodebook = hipCreateChannelDesc<FlexibleCodebookType>();
    checkCudaErrors(hipMalloc3DArray(&d_flexibleCodebookArray, &channelDescFlexibleCodebook, flexibleVolumeSize));

    // create 3D array for flexibleErrorsbook
    hipChannelFormatDesc channelDescFlexibleErrorsbook = hipCreateChannelDesc<FlexibleErrorsbookType>();
    checkCudaErrors(hipMalloc3DArray(&d_flexibleErrorsbookArray, &channelDescFlexibleErrorsbook, flexibleHistogramSize, hipArrayLayered));

    // create 3D array for simpleSpanLow
    hipChannelFormatDesc channelDescSimpleSpanLow = hipCreateChannelDesc<SpanType>();
    checkCudaErrors(hipMalloc3DArray(&d_simpleSpanLowArray, &channelDescSimpleSpanLow, flexibleVolumeSize));

    // create 3D array for simpleSpanHigh
    hipChannelFormatDesc channelDescSimpleSpanHigh = hipCreateChannelDesc<SpanType>();
    checkCudaErrors(hipMalloc3DArray(&d_simpleSpanHighArray, &channelDescSimpleSpanHigh, flexibleVolumeSize));

    // create 3D array for simpleCount
    hipChannelFormatDesc channelDescSimpleCount = hipCreateChannelDesc<SimpleCountType>();
    checkCudaErrors(hipMalloc3DArray(&d_simpleCountArray, &channelDescSimpleCount, flexibleVolumeSize));

    // create 3D array for simpleHistogram
    hipChannelFormatDesc channelDescSimpleHistogram = hipCreateChannelDesc<SimpleHistogramType>();
    checkCudaErrors(hipMalloc3DArray(&d_simpleHistogramArray, &channelDescSimpleHistogram, flexibleHistogramSize, hipArrayLayered));

    // create 3D array for flexibleTemplates
	hipChannelFormatDesc channelDescFlexibleTemplates = hipCreateChannelDesc<TemplatesType>();
    checkCudaErrors(hipMalloc3DArray(&d_flexibleTemplatesArray, &channelDescFlexibleTemplates, flexibleTemplatesSize, hipArrayLayered));

	////////////////////////////////////////////////
	// flexible block size
	///////////////////////////////////////////////

	// copy data to 3D array of codebookSpanLow
	hipMemcpy3DParms copyParamsCodebookSpanLow = {0};
	copyParamsCodebookSpanLow.srcPtr	= make_hipPitchedPtr(h_codebookSpanLow, flexibleVolumeSize.width*sizeof(SpanType), flexibleVolumeSize.width, flexibleVolumeSize.height);
	copyParamsCodebookSpanLow.dstArray	= d_codebookSpanLowArray;
	copyParamsCodebookSpanLow.extent	= flexibleVolumeSize;
	copyParamsCodebookSpanLow.kind		= hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParamsCodebookSpanLow));

	// copy data to 3D array of codebookSpanHigh
	hipMemcpy3DParms copyParamsCodebookSpanHigh = {0};
	copyParamsCodebookSpanHigh.srcPtr	= make_hipPitchedPtr(h_codebookSpanHigh, flexibleVolumeSize.width*sizeof(SpanType), flexibleVolumeSize.width, flexibleVolumeSize.height);
	copyParamsCodebookSpanHigh.dstArray	= d_codebookSpanHighArray;
	copyParamsCodebookSpanHigh.extent	= flexibleVolumeSize;
	copyParamsCodebookSpanHigh.kind		= hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParamsCodebookSpanHigh));

	// copy data to 3D array of flexibleCodebook
	hipMemcpy3DParms copyParamsFlexibleCodebook = {0};
	copyParamsFlexibleCodebook.srcPtr	= make_hipPitchedPtr(h_flexibleCodebook, flexibleVolumeSize.width*sizeof(FlexibleCodebookType), flexibleVolumeSize.width, flexibleVolumeSize.height);
	copyParamsFlexibleCodebook.dstArray	= d_flexibleCodebookArray;
	copyParamsFlexibleCodebook.extent	= flexibleVolumeSize;
	copyParamsFlexibleCodebook.kind		= hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParamsFlexibleCodebook));

	// copy data to 3D array of flexibleErrorsbook
	hipMemcpy3DParms copyParamsFlexibleErrorsbook = {0};
	copyParamsFlexibleErrorsbook.srcPtr 	= make_hipPitchedPtr(h_flexibleErrorsbook, flexibleHistogramSize.width*sizeof(FlexibleErrorsbookType), flexibleHistogramSize.width, flexibleHistogramSize.height);
	copyParamsFlexibleErrorsbook.dstArray	= d_flexibleErrorsbookArray;
	copyParamsFlexibleErrorsbook.extent		= flexibleHistogramSize;
	copyParamsFlexibleErrorsbook.kind		= hipMemcpyHostToDevice;

	checkCudaErrors(hipMemcpy3D(&copyParamsFlexibleErrorsbook));

	// copy data to 3D array of simpleLow
	hipMemcpy3DParms copyParamsSimpleLow = {0};
	copyParamsSimpleLow.srcPtr		= make_hipPitchedPtr(h_simpleLow, flexibleVolumeSize.width*sizeof(SpanType), flexibleVolumeSize.width, flexibleVolumeSize.height);
	copyParamsSimpleLow.dstArray	= d_simpleSpanLowArray;
	copyParamsSimpleLow.extent		= flexibleVolumeSize;
	copyParamsSimpleLow.kind		= hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParamsSimpleLow));

	// copy data to 3D array of simpleHigh
	hipMemcpy3DParms copyParamsSimpleHigh = {0};
	copyParamsSimpleHigh.srcPtr		= make_hipPitchedPtr(h_simpleHigh, flexibleVolumeSize.width*sizeof(SpanType), flexibleVolumeSize.width, flexibleVolumeSize.height);
	copyParamsSimpleHigh.dstArray	= d_simpleSpanHighArray;
	copyParamsSimpleHigh.extent		= flexibleVolumeSize;
	copyParamsSimpleHigh.kind		= hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParamsSimpleHigh));

	// copy data to 3D array of simpleCount
	hipMemcpy3DParms copyParamsSimpleCount = {0};
	copyParamsSimpleCount.srcPtr	= make_hipPitchedPtr(h_simpleCount, flexibleVolumeSize.width*sizeof(SimpleCountType), flexibleVolumeSize.width, flexibleVolumeSize.height);
	copyParamsSimpleCount.dstArray	= d_simpleCountArray;
	copyParamsSimpleCount.extent	= flexibleVolumeSize;
	copyParamsSimpleCount.kind		= hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParamsSimpleCount));

	// copy data to 3D array of simpleHistogram
	hipMemcpy3DParms copyParamsSimpleHistogram = {0};
	copyParamsSimpleCount.srcPtr	= make_hipPitchedPtr(h_simpleHistogram, flexibleHistogramSize.width*sizeof(SimpleHistogramType), flexibleHistogramSize.width, flexibleHistogramSize.height);
	copyParamsSimpleCount.dstArray	= d_simpleHistogramArray;
	copyParamsSimpleCount.extent	= flexibleHistogramSize;
	copyParamsSimpleCount.kind		= hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParamsSimpleCount));

	// copy data to 3D array of flexibleTemplates
	hipMemcpy3DParms copyParamsFlexibleTemplates = {0};
	copyParamsFlexibleTemplates.srcPtr		= make_hipPitchedPtr(h_flexibleTemplates, flexibleTemplatesSize.width*sizeof(TemplatesType), flexibleTemplatesSize.width, flexibleTemplatesSize.height);
	copyParamsFlexibleTemplates.dstArray	= d_flexibleTemplatesArray;
	copyParamsFlexibleTemplates.extent		= flexibleTemplatesSize;
	copyParamsFlexibleTemplates.kind		= hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParamsFlexibleTemplates));

	///////////////////////////////////////////////
	// flexible block size
	///////////////////////////////////////////////

	// set texture parameters for codebookSpanLowTex
	codebookSpanLowTex.normalized = false;
	codebookSpanLowTex.filterMode = hipFilterModePoint;
	codebookSpanLowTex.addressMode[0] = hipAddressModeClamp;
	codebookSpanLowTex.addressMode[1] = hipAddressModeClamp;

	// set texture parameters for codebookSpanHighTex
	codebookSpanHighTex.normalized = false;
	codebookSpanHighTex.filterMode = hipFilterModePoint;
	codebookSpanHighTex.addressMode[0] = hipAddressModeClamp;
	codebookSpanHighTex.addressMode[1] = hipAddressModeClamp;

	// set texture parameters for flexibleCodebookTex
	flexibleCodebookTex.normalized = false;
	flexibleCodebookTex.filterMode = hipFilterModePoint;
	flexibleCodebookTex.addressMode[0] = hipAddressModeClamp;
	flexibleCodebookTex.addressMode[1] = hipAddressModeClamp;

	// set texture parameters for flexibleErrorsbookTex
	flexibleErrorsbookTex.normalized = false;
	flexibleErrorsbookTex.filterMode = hipFilterModePoint;
	flexibleErrorsbookTex.addressMode[0] = hipAddressModeClamp;
	flexibleErrorsbookTex.addressMode[1] = hipAddressModeClamp;

	// set texture parameters for simpleSpanLowTex
	simpleSpanLowTex.normalized = false;
	simpleSpanLowTex.filterMode = hipFilterModePoint;
	simpleSpanLowTex.addressMode[0] = hipAddressModeClamp;
	simpleSpanLowTex.addressMode[1] = hipAddressModeClamp;

	// set texture parameters for simpleSpanHighTex
	simpleSpanHighTex.normalized = false;
	simpleSpanHighTex.filterMode = hipFilterModePoint;
	simpleSpanHighTex.addressMode[0] = hipAddressModeClamp;
	simpleSpanHighTex.addressMode[1] = hipAddressModeClamp;

	// set texture parameters for simpleCountTex
	simpleCountTex.normalized = false;
	simpleCountTex.filterMode = hipFilterModePoint;
	simpleCountTex.addressMode[0] = hipAddressModeClamp;
	simpleCountTex.addressMode[1] = hipAddressModeClamp;

	// set texture parameters for simpleHistogramTex
	simpleHistogramTex.normalized = false;
	simpleHistogramTex.filterMode = hipFilterModePoint;
	simpleHistogramTex.addressMode[0] = hipAddressModeClamp;
	simpleHistogramTex.addressMode[1] = hipAddressModeClamp;

	// set texture parameters for flexibleTemplatesTex
	flexibleTemplatesTex.normalized = false;
	flexibleTemplatesTex.filterMode = hipFilterModePoint;
	flexibleTemplatesTex.addressMode[0] = hipAddressModeClamp;
	flexibleTemplatesTex.addressMode[1] = hipAddressModeClamp;

	/////////////////////////////////////////////////////////////
	// flexible block size
	/////////////////////////////////////////////////////////////

	// bind array to 3D texture of codebookSpanLow
	checkCudaErrors(hipBindTextureToArray(codebookSpanLowTex, d_codebookSpanLowArray, channelDescCodebookSpanLow));

	// bind array to 3D texture of codebookSpanHigh
	checkCudaErrors(hipBindTextureToArray(codebookSpanHighTex, d_codebookSpanHighArray, channelDescCodebookSpanHigh));

	// bind array to 3D texture of flexibleCodebook
	checkCudaErrors(hipBindTextureToArray(flexibleCodebookTex, d_flexibleCodebookArray, channelDescFlexibleCodebook));

	// bind array to 2D layered texture of flexibleErrorsbook
	checkCudaErrors(hipBindTextureToArray(flexibleErrorsbookTex, d_flexibleErrorsbookArray, channelDescFlexibleErrorsbook));

	// bind array to 3D texture of simpleSpanLow
	checkCudaErrors(hipBindTextureToArray(simpleSpanLowTex, d_simpleSpanLowArray, channelDescSimpleSpanLow));

	// bind array to 3D texture of simpleSpanHigh
	checkCudaErrors(hipBindTextureToArray(simpleSpanHighTex, d_simpleSpanHighArray, channelDescSimpleSpanHigh));

	// bind array to 3D texture of simpleCount
	checkCudaErrors(hipBindTextureToArray(simpleCountTex, d_simpleCountArray, channelDescSimpleCount));

	// bind array to 2D layered texture of simpleHistogram
	checkCudaErrors(hipBindTextureToArray(simpleHistogramTex, d_simpleHistogramArray, channelDescSimpleHistogram));

	// bind array to 2D layered texture of flexibleTemplates
	checkCudaErrors(hipBindTextureToArray(flexibleTemplatesTex, d_flexibleTemplatesArray, channelDescFlexibleTemplates));

	//for (int i = 0; i < 622; i++) {
	//	for (int j = 0; j < flexNBin; j++) {
	//		if (h_templates[i * flexNBin + j] < 0 || h_templates[i * flexNBin + j] > 1) {
	//			printf("ERROR! template(%d, %d): %f\n", i, j, h_templates[i * flexNBin + j]);
	//		}
	//		if (i == 32) {
	//			//printf("template(32, %d): %f\n", j, h_templates[i * nBins + j]);
	//		}
	//	}
	//}

}// end function

void freeCudaBuffers()
{
	// flexible block size
	checkCudaErrors(hipFreeArray(d_codebookSpanLowArray));
	checkCudaErrors(hipFreeArray(d_codebookSpanHighArray));
	checkCudaErrors(hipFreeArray(d_flexibleCodebookArray));
	checkCudaErrors(hipFreeArray(d_flexibleErrorsbookArray));
	checkCudaErrors(hipFreeArray(d_simpleSpanLowArray));
	checkCudaErrors(hipFreeArray(d_simpleSpanHighArray));
	checkCudaErrors(hipFreeArray(d_simpleCountArray));
	checkCudaErrors(hipFreeArray(d_simpleHistogramArray));
	checkCudaErrors(hipFreeArray(d_flexibleTemplatesArray));

	checkCudaErrors(hipFreeArray(d_flexTexArray));
}
