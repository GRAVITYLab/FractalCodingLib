#include "hip/hip_runtime.h"
#include "FEL_core_gpu.h"

void
kernel_wrapper( double** rangeData, double** domainData, int nRange, int nDomain, int nBin )
{
	int* matchedDomainIdList = new int[nRange];
	int* matchedDomainIdList_d = 0;
	double* rangeData_d = 0;
	double* domainData_d = 0;
	double* latestErrorList_d = 0;
	memset( matchedDomainIdList, 0, sizeof(int)*nRange );

	// Initialize threads and blocks
	dim3 threads( 16, 1 );
	dim3 blocks( 16, 12 );

	// Allocate device memory
	//CUDA_SAFE_CALL( 
			hipMalloc( (void **)&rangeData_d, nRange*nBin*sizeof(double) );// );
	//CUDA_SAFE_CALL( 
			hipMalloc( (void **)&domainData_d, nDomain*nBin*sizeof(double) );// );
	//CUDA_SAFE_CALL( 
			hipMalloc( (void **)&matchedDomainIdList_d, nRange*sizeof(int) );// );
	//CUDA_SAFE_CALL( 
			hipMalloc( (void **)&latestErrorList_d, nRange*sizeof(double) );// );

	// Copy data to device memory
	//CUDA_SAFE_CALL( 
			hipMemcpy( rangeData_d, (*rangeData), nRange*nBin*sizeof(double), hipMemcpyHostToDevice );// );
	//CUDA_SAFE_CALL( 
			hipMemcpy( domainData_d, (*domainData), nDomain*nBin*sizeof(double), hipMemcpyHostToDevice );// );

	// Launch encoding kernel
	int nDomPerRound = 80;
	int nRound = nDomain / nDomPerRound;
	fprintf( stderr, "Number of rounds needed: %d\n", nRound );

	double gpuTime;
	unsigned int hTimer;
	//CUT_SAFE_CALL( 
			//cutCreateTimer(&hTimer) ;//);
	//CUT_SAFE_CALL( 
			//cutResetTimer(hTimer) ;//);
	//CUT_SAFE_CALL( 
			//cutStartTimer(hTimer) ;//);

	for( int iR = 0; iR<nRound; iR ++ )
	{
		fprintf( stderr, "Kernel call round: %d\n", iR );
		encode<<<blocks, 1>>>( rangeData_d, domainData_d,
							   matchedDomainIdList_d,
							   latestErrorList_d,
							   nRange, nDomain, nBin,
							   iR*nDomPerRound );
	}

	//CUT_CHECK_ERROR("encode() execution failed\n");
	//CUDA_SAFE_CALL( 
		hipDeviceSynchronize() ;//);

	//CUT_SAFE_CALL( 
			//cutStopTimer(hTimer) ;//);
	//gpuTime = cutGetTimerValue(hTimer);

	// Copy data back to device memory
	//CUDA_SAFE_CALL( 
		hipMemcpy( matchedDomainIdList, matchedDomainIdList_d, nRange*sizeof(int), hipMemcpyDeviceToHost ) ;//);

	// Deallocate device memory
	//CUDA_SAFE_CALL( 
			hipFree( latestErrorList_d ) ;//);
	//CUDA_SAFE_CALL( 
			hipFree( matchedDomainIdList_d ) ;//);
	//CUDA_SAFE_CALL( 
			hipFree( rangeData_d ) ;//);
	//CUDA_SAFE_CALL( 
			hipFree( domainData_d ) ;//);

	fprintf( stderr, "Time: %g milliseconds\n", gpuTime );

	//for( int i=0; i<nRange; i++ )
	//	fprintf( stderr, "%d\n", matchedDomainIdList[i] );

	delete [] matchedDomainIdList;

}// end function

/*
void
kernel_wrapper_incremental_1( double** domainData, int nDomain, int nBin )
{
	double* domainData_d = 0;
	
	// Allocate device memory
	CUDA_SAFE_CALL( hipMalloc( (void **)&domainData_d, nDomain*nBin*sizeof(double) ) );
	
}// end function

void
kernel_wrapper_incremental_2( double** rangeData, int nBin )
{
	int* matchedDomainIdList = new int[nRange];
	int* matchedDomainIdList_d = 0;
	double* rangeData_d = 0;

	double* latestErrorList_d = 0;
	memset( matchedDomainIdList, 0, sizeof(int)*nRange );

	// Initialize threads and blocks
	dim3 threads( 16, 1 );
	dim3 blocks( 16, 12 );

	// Allocate device memory
	CUDA_SAFE_CALL( hipMalloc( (void **)&rangeData_d, nRange*nBin*sizeof(double) ) );
	CUDA_SAFE_CALL( hipMalloc( (void **)&matchedDomainIdList_d, nRange*sizeof(int) ) );
	CUDA_SAFE_CALL( hipMalloc( (void **)&latestErrorList_d, nRange*sizeof(double) ) );

	// Copy data to device memory
	CUDA_SAFE_CALL( hipMemcpy( rangeData_d, (*rangeData), nRange*nBin*sizeof(double), hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy( domainData_d, (*domainData), nDomain*nBin*sizeof(double), hipMemcpyHostToDevice ) );

	// Launch encoding kernel
	int nDomPerRound = 80;
	int nRound = nDomain / nDomPerRound;
	fprintf( stderr, "Number of rounds needed: %d\n", nRound );

	double gpuTime;
	unsigned int hTimer;
	CUT_SAFE_CALL( cutCreateTimer(&hTimer) );
	CUT_SAFE_CALL( cutResetTimer(hTimer) );
	CUT_SAFE_CALL( cutStartTimer(hTimer) );

	for( int iR = 0; iR<nRound; iR ++ )
	{
		fprintf( stderr, "Kernel call round: %d\n", iR );
		encode<<<blocks, 1>>>( rangeData_d, domainData_d,
							   matchedDomainIdList_d,
							   latestErrorList_d,
							   nRange, nDomain, nBin,
							   iR*nDomPerRound );
	}

	CUT_CHECK_ERROR("encode() execution failed\n");
	CUDA_SAFE_CALL( hipDeviceSynchronize() );

	CUT_SAFE_CALL( cutStopTimer(hTimer) );
	gpuTime = cutGetTimerValue(hTimer);

	// Copy data back to device memory
	CUDA_SAFE_CALL( hipMemcpy( matchedDomainIdList, matchedDomainIdList_d, nRange*sizeof(int), hipMemcpyDeviceToHost ) );

	// Deallocate device memory
	CUDA_SAFE_CALL( hipFree( latestErrorList_d ) );
	CUDA_SAFE_CALL( hipFree( matchedDomainIdList_d ) );
	CUDA_SAFE_CALL( hipFree( rangeData_d ) );
	CUDA_SAFE_CALL( hipFree( domainData_d ) );

	fprintf( stderr, "Time: %g milliseconds\n", gpuTime );

	//for( int i=0; i<nRange; i++ )
	//	fprintf( stderr, "%d\n", matchedDomainIdList[i] );

	delete [] matchedDomainIdList;

}// end function
*/

__global__ void
encode( double* rangeData_d, double* domainData_d,
		int* matchedDomainIdList_d,
		double* latestErrorList_d,
		int nRange, int nDomain, int nBin,
		int domStartID )
{
	int tid = (blockIdx.y * gridDim.x * gridDim.y) + blockIdx.x * gridDim.x + threadIdx.x;

	int bestMatchDomainId;
	double buffer, matchErrorPrcnt, minErrorPrcnt;
	double curRange[130];
	double domainFreqList[130];
	double domainFreqListCopy[130];
	double domainFreqListRev[130];
	double* buffer2[130];

	if( tid < nRange )
	{
		memcpy( curRange, rangeData_d + tid*nBin, sizeof(double)*nBin );

		if( domStartID == 0 )
			minErrorPrcnt = 1000;
		else
			minErrorPrcnt = latestErrorList_d[tid];
		bestMatchDomainId = 0;

		int domMaxId = min ( domStartID+10, nDomain );
		for( int iD = domStartID; iD<domMaxId; iD++ )
		{
			memcpy( domainFreqList, domainData_d + iD*nBin, sizeof(double)*nBin );

			// Circular shift
			memcpy( domainFreqListCopy, domainFreqList, sizeof(double)*nBin );
			for( int iT = 0; iT<nBin; iT++ )
			{
				// Place rightmost to buffer
				buffer = domainFreqListCopy[nBin-1];

				// Move everybody else by 1 bit
				memcpy( buffer2, domainFreqListCopy, sizeof(double)*(nBin-1) );
				memcpy( domainFreqListCopy+1, buffer2, sizeof(double)*(nBin-1) );

				// Fill back the leftmost
				domainFreqListCopy[0] = buffer;

				// Compare with range distribution
				matchErrorPrcnt = 0;
				for( int iB = 0; iB<nBin; iB++ )
				{
					double del = ( domainFreqListCopy[iB] - curRange[iB] );
					matchErrorPrcnt += del * del;
				}

				if( matchErrorPrcnt < minErrorPrcnt )
				{
					minErrorPrcnt = matchErrorPrcnt;
					bestMatchDomainId = iD;
					//optRot = iT;
					//optRef = false;
				}
			}

			// Reflect distribution
			for( int i=0; i<nBin; i++ )
				domainFreqListRev[i] = domainFreqList[nBin-i-1];

			// Circular shift
			memcpy( domainFreqListCopy, domainFreqListRev, sizeof(double)*nBin );
			for( int iT = 0; iT<nBin; iT++ )
			{
				// Place rightmost to buffer
				buffer = domainFreqListCopy[nBin-1];

				// Move everybody else by 1 bit
				memcpy( buffer2, domainFreqListCopy, sizeof(double)*(nBin-1) );
				memcpy( domainFreqListCopy+1, buffer2, sizeof(double)*(nBin-1) );

				// Fill back the leftmost
				domainFreqListCopy[0] = buffer;


				// Compare with range distribution
				matchErrorPrcnt = 0;
				for( int iB = 0; iB<nBin; iB++ )
				{
					double del = ( domainFreqListCopy[iB] - curRange[iB] );
					matchErrorPrcnt += del * del;
				}

				if( matchErrorPrcnt < minErrorPrcnt )
				{
					minErrorPrcnt = matchErrorPrcnt;
					bestMatchDomainId = iD;
					//optRot = iT;
					//optRef = true;
				}
			}
			//(*rotateAmount) = optRot;
			//(*isReflected) = optRef;

		}// end for : scan domains

		matchedDomainIdList_d[tid] = bestMatchDomainId;
		latestErrorList_d[tid] = minErrorPrcnt;

	}// end if (tid < nRange)

}// end function

/*
__global__ void
encode_incremental( double* rangeData_d, double* domainData_d,
					int* matchedDomainIdList_d,
					double* latestErrorList_d,
					int nRange, int nDomain, int nBin,
					int domStartID )
{
	// Declare shared memory which stores 
	// comparison for one range
	double* errorList = new double[nDomain];	
	
	// Each thread can perform one comparison
	int tid = blockIdx.y * gridDim.x * gridDim.y) + blockIdx.x * gridDim.x + threadIdx.x;

	int bestMatchDomainId;
	double buffer, matchErrorPrcnt, minErrorPrcnt;
	double curRange[130];
	double domainFreqList[130];
	double domainFreqListCopy[130];
	double domainFreqListRev[130];
	double* buffer2[130];

	if( tid < nRange )
	{
		memcpy( curRange, rangeData_d + tid*nBin, sizeof(double)*nBin );

		if( domStartID == 0 )
			minErrorPrcnt = 1000;
		else
			minErrorPrcnt = latestErrorList_d[tid];
		bestMatchDomainId = 0;

		int domMaxId = min ( domStartID+10, nDomain );
		for( int iD = domStartID; iD<domMaxId; iD++ )
		{
			memcpy( domainFreqList, domainData_d + iD*nBin, sizeof(double)*nBin );

			// Circular shift
			memcpy( domainFreqListCopy, domainFreqList, sizeof(double)*nBin );
			for( int iT = 0; iT<nBin; iT++ )
			{
				// Place rightmost to buffer
				buffer = domainFreqListCopy[nBin-1];

				// Move everybody else by 1 bit
				memcpy( buffer2, domainFreqListCopy, sizeof(double)*(nBin-1) );
				memcpy( domainFreqListCopy+1, buffer2, sizeof(double)*(nBin-1) );

				// Fill back the leftmost
				domainFreqListCopy[0] = buffer;

				// Compare with range distribution
				matchErrorPrcnt = 0;
				for( int iB = 0; iB<nBin; iB++ )
				{
					double del = ( domainFreqListCopy[iB] - curRange[iB] );
					matchErrorPrcnt += del * del;
				}

				if( matchErrorPrcnt < minErrorPrcnt )
				{
					minErrorPrcnt = matchErrorPrcnt;
					bestMatchDomainId = iD;
					//optRot = iT;
					//optRef = false;
				}
			}

			// Reflect distribution
			for( int i=0; i<nBin; i++ )
				domainFreqListRev[i] = domainFreqList[nBin-i-1];

			// Circular shift
			memcpy( domainFreqListCopy, domainFreqListRev, sizeof(double)*nBin );
			for( int iT = 0; iT<nBin; iT++ )
			{
				// Place rightmost to buffer
				buffer = domainFreqListCopy[nBin-1];

				// Move everybody else by 1 bit
				memcpy( buffer2, domainFreqListCopy, sizeof(double)*(nBin-1) );
				memcpy( domainFreqListCopy+1, buffer2, sizeof(double)*(nBin-1) );

				// Fill back the leftmost
				domainFreqListCopy[0] = buffer;


				// Compare with range distribution
				matchErrorPrcnt = 0;
				for( int iB = 0; iB<nBin; iB++ )
				{
					double del = ( domainFreqListCopy[iB] - curRange[iB] );
					matchErrorPrcnt += del * del;
				}

				if( matchErrorPrcnt < minErrorPrcnt )
				{
					minErrorPrcnt = matchErrorPrcnt;
					bestMatchDomainId = iD;
					//optRot = iT;
					//optRef = true;
				}
			}
			//(*rotateAmount) = optRot;
			//(*isReflected) = optRef;

		}// end for : scan domains

		matchedDomainIdList_d[tid] = bestMatchDomainId;
		latestErrorList_d[tid] = minErrorPrcnt;

	}// end if (tid < nRange)

}// end function
*/

__global__ void
decode( float *pDataA, float *pDataB, float *pResult)
{

}// end function
